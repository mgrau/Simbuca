#include "hip/hip_runtime.h"
/*
NBODY algorithm from NVIDIA SDK. 
Changes:
1) bodyBodyInteraction function computes Coulomb interaction without the factor qq/(4 pi eps0)
2) gravitation function, solving a problem with multithreadBodies ON 
*/


#include <math.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

__constant__ float softeningSquared = 0.0000001f;

struct SharedMemory
{
    __device__ inline operator       float4*()
    {
        extern __shared__ int __smem[];
        return (float4*)__smem;
    }

    __device__ inline operator const float4*() const
    {
        extern __shared__ int __smem[];
        return (float4*)__smem;
    }
};


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) 
{
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}




int nbody;
// body value arrays (pos + mass, velocity)
//float4 *p_m;
//float3 *acc;


// body value arrays for device
float4 *p_m_Dev;
float3 *acc_Dev;

//CUDA timing
//hipEvent_t start, stop;

//CUDA PROPS
int p_nb = 512;
int q = 1;
bool bMT =false;
dim3 grid, threads;
int sharedMemSize;


void copyDataToDevice_Async(float4* h_p)
{
	HANDLE_ERROR (hipMemcpyAsync(p_m_Dev, h_p, nbody * sizeof (float4), hipMemcpyHostToDevice));
}

void copyDataToDevice_Sync(float4* h_p)
{
	HANDLE_ERROR (hipMemcpy(p_m_Dev, h_p, nbody * sizeof (float4), hipMemcpyHostToDevice));
}



void copyDeviceToHost_Async(float3 * res)
{	
	hipMemcpyAsync (res, acc_Dev, nbody * sizeof (float3), hipMemcpyDeviceToHost);
}

void copyDeviceToHost_Sync(float3 * res)
{	
	hipMemcpy (res, acc_Dev, nbody * sizeof (float3), hipMemcpyDeviceToHost);		
}

void freeDevice(void)
{
	hipFree (p_m_Dev);
	hipFree (acc_Dev);
	
}

void initDataToDevice(void)
{
	HANDLE_ERROR (hipMalloc ((void **) &p_m_Dev, nbody * sizeof (float4)));
	HANDLE_ERROR (hipMalloc ((void **) &acc_Dev, nbody * sizeof (float3)));
}



extern "C" void initialize(int n_,int p_, int q_ ,int gridx_)
{		
 	nbody= n_;
	//p_m = (float4 *) malloc (nbody * sizeof (float4));
	//acc = (float3 *) malloc (nbody * sizeof (float3));
	
	
 	initDataToDevice();
	
	hipDeviceProp_t props;
	
	if(p_*q_*gridx_==0)
	{
		hipGetDeviceProperties(&props, 0);
	
		p_nb=min(nbody,p_nb);
		
		while ((nbody > 0) && p_nb > 1 && (nbody / p_nb < int((unsigned)props.multiProcessorCount)))
       		{
            		p_nb /= 2;
            		q *= 2;
        	}

       		grid.x = (int)(nbody + (p_nb-1))/p_nb;
	}
	else
	{
		p_nb = p_;
		q = q_;
		grid.x = gridx_;
	}
	
	threads.x = p_nb;
	threads.y = q;
	threads.z = 1;
	grid.y = 1;
	grid.z = 1;
	printf("######################################\n");
	printf("            NBODY ALGO USED         \n");
	printf(" tiles of %d x %d bodies              \n",p_nb,p_nb);
	printf(" %d thread(s) per body               \n",q);
	printf(" %d blocks used                       \n",grid.x);
	printf("######################################\n");
	
	if (grid.x > 0 && threads.y == 1)
        {
          	bMT = false;
	  	
        }
       	else if (grid.x > 0)
       	{
          	bMT = true;
	}
	
	sharedMemSize = p_nb * q  * 4 * sizeof(float);	
}


extern "C" float finalize(void)
{
	float elapsed=0;
	freeDevice();
	return elapsed;
}


__device__  float3 bodyBodyInteraction(float3  ai, float4 bi, float4 bj) 
{
   float3 r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += softeningSquared;

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    float invDist = rsqrt(distSqr);
    float invDistCube =  (invDist * invDist * invDist);

    // s = m_j * invDistCube [1 FLOP]
    // float s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    /*
    ai.x += r.x*s;// * s;
    ai.y += r.y*s;// * s;
    ai.z += r.z*s;// * s;
    */
    
    ai.x += r.x*invDistCube;// * s;
    ai.y += r.y*invDistCube;// * s;
    ai.z += r.z*invDistCube;// * s;
    
    return ai;
}


__device__ float3 gravitation(float4 iPos, float3  accel)
{
    float4 * sharedPos = SharedMemory();
    //extern __shared__ float4 sharedPos[];
    // The CUDA 1.1 compiler cannot determine that i is not going to 
    // overflow in the loop below.  Therefore if int is used on 64-bit linux 
    // or windows (or long instead of long long on win64), the compiler
    // generates suboptimal code.  Therefore we use long long on win64 and
    // long on everything else. (Workaround for Bug ID 347697)

    unsigned long j = 0;


    // Here we unroll the loop to reduce bookkeeping instruction overhead
    // 32x unrolling seems to provide best performance

    // Note that having an unsigned int loop counter and an unsigned
    // long index helps the compiler generate efficient code on 64-bit
    // OSes.  The compiler can't assume the 64-bit index won't overflow
    // so it incurs extra integer operations.  This is a standard issue
    // in porting 32-bit code to 64-bit OSes.

#pragma unroll 32
    for (unsigned int counter = 0; counter < blockDim.x; counter++ ) 
    {
        accel = bodyBodyInteraction(accel, iPos, SX(j++)); 
    }

    return accel;
}

// WRAP is used to force each block to start working on a different 
// chunk (and wrap around back to the beginning of the array) so that
// not all multiprocessors try to read the same memory locations at 
// once.
#define WRAP(x,m) (((x)<m)?(x):(x-m))  // Mod without divide, works on values from 0 up to 2m

template <bool multithreadBodies>
__global__ void computeBodyAccel(float4 * positions, float3 * DevA, int numBodies)
{
    float4* sharedPos = SharedMemory();
    //extern __shared__ float4 sharedPos[];
    float3 acc = {0.0f, 0.0f, 0.0f};

    int p_nb = blockDim.x;
    int q = blockDim.y;
    int n = numBodies;
    int numTiles = n / (p_nb * q);
   // float3 * globalA = (float3*) DevA;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float4 myposition = positions[index];
 

    for (int tile = blockIdx.y; tile < numTiles + blockIdx.y; tile++) 
    {
        sharedPos[threadIdx.x+blockDim.x*threadIdx.y] = 
            multithreadBodies ? 
            positions[WRAP(blockIdx.x + q * tile + threadIdx.y, gridDim.x) * p_nb + threadIdx.x] :
        positions[WRAP(blockIdx.x + tile,                   gridDim.x) * p_nb + threadIdx.x];

        __syncthreads();

        // This is the "tile_calculation" function from the GPUG3 article.
        acc = gravitation(myposition, acc);

        __syncthreads();
    }
	
    // When the numBodies / thread block size is < # multiprocessors (16 on G80), the GPU is 
    // underutilized.  For example, with a 256 threads per block and 1024 bodies, there will only 
    // be 4 thread blocks, so the GPU will only be 25% utilized. To improve this, we use multiple 
    // threads per body.  We still can use blocks of 256 threads, but they are arranged in q rows 
    // of p threads each.  Each thread processes 1/q of the forces that affect each body, and then 
    // 1/q of the threads (those with threadIdx.y==0) add up the partial sums from the other 
    // threads for that body.  To enable this, use the "--p=" and "--q=" command line options to 
    // this example. e.g.: "nbody.exe --n=1024 --p=64 --q=4" will use 4 threads per body and 256 
    // threads per block. There will be n/p = 16 blocks, so a G80 GPU will be 100% utilized.

    // We use a bool template parameter to specify when the number of threads per body is greater 
    // than one, so that when it is not we don't have to execute the more complex code required!
    if (multithreadBodies)
    {
        SX_SUM(threadIdx.x, threadIdx.y).x = acc.x;
        SX_SUM(threadIdx.x, threadIdx.y).y = acc.y;
        SX_SUM(threadIdx.x, threadIdx.y).z = acc.z;
        __syncthreads();
        // Save the result in global memory for the integration step
       // if (threadIdx.y == 0) 
        {
	
	acc.z =0.0f;
	acc.x =0.0f;
	acc.y =0.0f;
	
            for (int i = 0; i < blockDim.y; i++) 
            {
                acc.x += SX_SUM(threadIdx.x,i).x;
                acc.y += SX_SUM(threadIdx.x,i).y;
                acc.z += SX_SUM(threadIdx.x,i).z;
            }
        }
	//__syncthreads();
    } 
    //float3 acc3 = {acc.x,acc.y,acc.z};
    // globalA[index] = acc3;
    DevA[index] =acc;
}

	
extern "C"  void simulation_Async(float4 * h_p,float3 * res)
	{
	copyDataToDevice_Async(h_p);
	
	if(bMT)
	computeBodyAccel<true><<< grid,threads,sharedMemSize,0 >>>(p_m_Dev,acc_Dev,nbody);
	else
	computeBodyAccel<false><<< grid,threads,sharedMemSize,0 >>>(p_m_Dev,acc_Dev,nbody);
	
	copyDeviceToHost_Async(res);
	return;
	}
	
extern "C"  void simulation_Sync(float4 * h_p,float3 * res)
	{
	copyDataToDevice_Sync(h_p);
	
	if(bMT)
	computeBodyAccel<true><<< grid,threads,sharedMemSize,0 >>>(p_m_Dev,acc_Dev,nbody);
	else
	computeBodyAccel<false><<< grid,threads,sharedMemSize,0 >>>(p_m_Dev,acc_Dev,nbody);
	
	copyDeviceToHost_Sync(res);
	return;
	}
